#include "hip/hip_runtime.h"
#include "integrator.h"
#include "cuda/kernel.h"
#include "cuda/util.h"

using namespace Pupil;

namespace wf {
    void InitialPath(uint2 launch_size, cuda::RWDataView<GlobalData>& g_data, cuda::Stream* stream) noexcept;
    void ScatterRays(unsigned int launch_size, cuda::RWDataView<GlobalData>& g_data, cuda::Stream* stream) noexcept;
    void AccumulateRadiance(unsigned int launch_size, cuda::RWDataView<GlobalData>& g_data, cuda::Stream* stream) noexcept;
}// namespace wf

namespace wf {
    void Integrator::Trace(cuda::RWDataView<GlobalData>& g_data, Pupil::cuda::Stream* stream) noexcept {
        Pupil::cuda::LaunchKernel([g_data] __device__() { g_data->depth = 0; g_data->path_record.Clear(); }, stream);
        InitialPath(m_frame_size, g_data, stream);

        for (int depth = 0; depth < m_max_depth; ++depth) {
            Pupil::cuda::LaunchKernel([g_data] __device__() { g_data->hit_record.Clear(); }, stream);

            m_ray_pass->Run(reinterpret_cast<hipDeviceptr_t>(g_data.GetDataPtr()), m_max_wave_size, 1);
            if (depth == m_max_depth - 1) break;

            Pupil::cuda::LaunchKernel([g_data] __device__() {  g_data->depth++; g_data->path_record.Clear(); g_data->nee_record.Clear(); }, stream);

            ScatterRays(m_max_wave_size, g_data, stream);
            m_shadow_ray_pass->Run(reinterpret_cast<hipDeviceptr_t>(g_data.GetDataPtr()), m_max_wave_size, 1);
        }
        AccumulateRadiance(m_max_wave_size, g_data, stream);
    }

    void InitialPath(uint2 launch_size, Pupil::cuda::RWDataView<GlobalData>& g_data, Pupil::cuda::Stream* stream) noexcept {
        Pupil::cuda::LaunchKernel2D(
            launch_size, [g_data] __device__(uint2 index, uint2 size) {
                const unsigned int pixel_index = index.y * size.x + index.x;
                auto&              camera      = g_data->camera;
                cuda::Random       random;
                random.Init(4, pixel_index, g_data->random_seed);

                const float2 subpixel_jitter = random.Next2();

                const float2 subpixel =
                    make_float2(
                        (static_cast<float>(index.x) + subpixel_jitter.x) / static_cast<float>(size.x),
                        (static_cast<float>(index.y) + subpixel_jitter.y) / static_cast<float>(size.y));
                // const float2 subpixel = make_float2((static_cast<float>(index.x)) / w, (static_cast<float>(index.y)) / h);
                const float4 point_on_film = make_float4(subpixel, 0.f, 1.f);

                float4 d = camera.sample_to_camera * point_on_film;

                d /= d.w;
                d.w = 0.f;
                d   = normalize(d);

                float3 ray_direction = normalize(make_float3(camera.camera_to_world * d));

                float3 ray_origin = make_float3(
                    camera.camera_to_world.r0.w,
                    camera.camera_to_world.r1.w,
                    camera.camera_to_world.r2.w);

                auto new_camera_ray_path = g_data->path_record.Alloc();
                new_camera_ray_path.ray_dir(ray_direction);
                new_camera_ray_path.ray_origin(ray_origin);
                new_camera_ray_path.throughput(make_float3(1.f, 1.f, 1.f));
                new_camera_ray_path.random_seed(random.GetSeed());
                new_camera_ray_path.pixel_index(pixel_index);

                g_data->frame_buffer[pixel_index] = make_float4(0.f, 0.f, 0.f, 1.f);
            },
            stream);
    }

    void ScatterRays(unsigned int launch_size, cuda::RWDataView<GlobalData>& g_data, cuda::Stream* stream) noexcept {
        Pupil::cuda::LaunchKernel1D(
            launch_size, [g_data] __device__(unsigned int launch_index, unsigned int size) {
                if (launch_index >= g_data->hit_record.GetNum()) return;
                auto       record      = g_data->hit_record[launch_index];
                const auto pixel_index = record.pixel_index();
                auto       ray_dir     = record.ray_dir();
                auto       throughput  = record.throughput();
                auto       bsdf        = record.mat();
                auto       geo         = record.geo();

                cuda::Random random;
                random.SetSeed(record.random_seed());

                // direct lighting
                {
                    auto emitter = g_data->emitters.SelectOneEmiiter(random.Next());

                    Pupil::optix::EmitterSampleRecord emitter_sample_record;
                    emitter->SampleDirect(emitter_sample_record, geo, random.Next2());

                    optix::BsdfSamplingRecord eval_record;
                    eval_record.wi      = optix::ToLocal(emitter_sample_record.wi, geo.normal);
                    eval_record.wo      = optix::ToLocal(-ray_dir, geo.normal);
                    eval_record.sampler = &random;
                    bsdf.Eval(eval_record);

                    float3 bsdf_eval_f   = eval_record.f;
                    float  bsdf_eval_pdf = eval_record.pdf;

                    float emit_pdf = emitter_sample_record.pdf * emitter->select_probability;
                    if (optix::IsValid(emit_pdf)) {
                        NEERecord nee;
                        nee.shadow_ray_dir    = emitter_sample_record.wi;
                        nee.shadow_ray_origin = geo.position;
                        nee.shadow_ray_t_max  = emitter_sample_record.distance - 0.0001f;

                        float NoL = abs(dot(geo.normal, emitter_sample_record.wi));
                        float mis = emitter_sample_record.is_delta ? 1.f : optix::MISWeight(emitter_sample_record.pdf, bsdf_eval_pdf);

                        nee.radiance = emitter_sample_record.radiance * throughput * bsdf_eval_f * NoL * mis / emit_pdf;

                        auto shadow_ray_record = g_data->nee_record.Alloc();
                        shadow_ray_record.shadow_ray_t_max(nee.shadow_ray_t_max);
                        shadow_ray_record.shadow_ray_dir(nee.shadow_ray_dir);
                        shadow_ray_record.shadow_ray_origin(nee.shadow_ray_origin);
                        shadow_ray_record.radiance(nee.radiance);
                        shadow_ray_record.pixel_index(pixel_index);
                    }
                }

                // bsdf sampling
                {
                    optix::BsdfSamplingRecord bsdf_sample_record;
                    bsdf_sample_record.wo      = optix::ToLocal(-ray_dir, geo.normal);
                    bsdf_sample_record.sampler = &random;
                    bsdf.Sample(bsdf_sample_record);

                    if (optix::IsValid(bsdf_sample_record.pdf)) {
                        PathRecord record;
                        record.bsdf_sample_pdf  = bsdf_sample_record.pdf;
                        record.bsdf_sample_type = bsdf_sample_record.sampled_type;
                        throughput *= bsdf_sample_record.f * abs(bsdf_sample_record.wi.z) / bsdf_sample_record.pdf;

                        record.ray_dir    = optix::ToWorld(bsdf_sample_record.wi, geo.normal);
                        record.ray_origin = geo.position;

                        auto next_path = g_data->path_record.Alloc();
                        next_path.ray_dir(record.ray_dir);
                        next_path.ray_origin(record.ray_origin);
                        next_path.throughput(throughput);
                        next_path.bsdf_sample_pdf(record.bsdf_sample_pdf);
                        next_path.bsdf_sample_type(record.bsdf_sample_type);
                        next_path.random_seed(random.GetSeed());
                        next_path.pixel_index(pixel_index);
                    }
                }
            },
            stream);
    }

    void AccumulateRadiance(unsigned int launch_size, cuda::RWDataView<GlobalData>& g_data, cuda::Stream* stream) noexcept {
        Pupil::cuda::LaunchKernel1D(
            launch_size, [g_data] __device__(unsigned int pixel_index, unsigned int size) {
                auto radiance = make_float3(g_data->frame_buffer[pixel_index]);

                if (g_data->config.accumulated_flag && g_data->sample_cnt > 0) {
                    const float  t   = 1.f / (g_data->sample_cnt + 1.f);
                    const float3 pre = make_float3(g_data->accum_buffer[pixel_index]);
                    radiance         = lerp(pre, radiance, t);
                }
                g_data->accum_buffer[pixel_index] = make_float4(radiance, 1.f);
                g_data->frame_buffer[pixel_index] = make_float4(radiance, 1.f);
            },
            stream);
    }
}// namespace wf